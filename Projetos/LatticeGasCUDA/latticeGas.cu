#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "rules.hpp"
#include "kernels.hpp"
#include "lattice.hpp"
#include "graphicsBase.hpp"

int UniformBetween(int nMax) {return rand()%nMax;}
using namespace std;

int main() {
	bool running = true;
	srand(time(NULL));
	Graphics::GraphicsBase2D graphics(900*sqrt(3.0)/2, 900, 0, 1.0, 0, sqrt(3.0)/2, 0.001, "Gas Lattice CUDA", true);
	Lattice lat(192*4,16);
	//Inicia todos os sites como zero
	for (int i=0; i<lat.N; i++) {
		lat.h_cells1[i]=0;
	}
	//Inicia o quadrado central
	for (int i=lat.L/4; i<3*lat.L/4; i++) {
		for (int j=lat.L/4; j<2*lat.L/4; j++) {
			lat.h_cells1[i*lat.L+j] = FULL; 
			lat.h_cells1[i*lat.L+j] = LEFT; 
		}
		for (int j=2*lat.L/4; j<3*lat.L/4; j++) {
			lat.h_cells1[i*lat.L+j] = FULL;
			lat.h_cells1[i*lat.L+j] = RIGHT; 
		}
	/*	for (int j=lat.L/4; j<3*lat.L/4; j++) {
			if (rand()/(double)RAND_MAX < .2)
				lat.h_cells1[i*lat.L+j] |= BARRIER; 
		}*/

	}

	lat.PrepareToOpenGL();
	lat.CopyToDevice();
	glewInit();
	
	//for(int i=0; i<1e2; i++) {
	while(true){
		graphics.Clear();
		lat.Draw();
		if (glfwGetKey('P') == GLFW_PRESS)
			running=true;
		if (running)
			for (int i=0; i<5;i++)
				lat.FullUpdate();
		graphics.EndFrame();
		hipDeviceSynchronize();
	}
}
